#include "hip/hip_runtime.h"
{
const real invR2 = invR*invR;
const real invR3 = invR2*invR;
const real invR4 = invR3*invR;
const real invR5 = invR4*invR;
const real invR6 = invR5*invR;
const real invR7 = invR6*invR;
const real invR8 = invR7*invR;
const real invR9 = invR8*invR;
const real invR10 = invR9*invR;

// const real r6 = r2*r2*r2;

const real d = sigmaEpsilon1.y + sigmaEpsilon2.y;
const real d2 = d*d;
const real d3 = d2*d;
const real d6 = d3*d3;

const real mdr = -d * r;

real expTerm = exp(mdr);

real c6Deriv = invR6 - expTerm * (
    invR6 * (mdr - 6) +
    d * invR5  * (mdr - 5) +
    d2 * 0.5 * invR4 * (mdr - 4) +
    d3 * 0.1666666667 * invR3 * (mdr - 3) +
    d2 * 0.04166666667 * invR2 * (mdr - 2) +
    d3 * d2 * 0.008333333333 * invR * (mdr - 1) +
    d6 * 0.001388888889 * mdr
);

real c8Deriv = invR8 - expTerm * (
    invR8 * (mdr - 8) +
    d * invR7 * (mdr - 7) +
    d2 * 0.5 * invR6 * (mdr - 6) +
    d3 * 0.1666666667 * invR5 * (mdr - 5) +
    d2 * d2 * 0.04166666667 * invR4 * (mdr - 4) +
    d3 * d2 * 0.008333333333 * invR3 * (mdr - 3) +
    d6 * 0.001388888889 * invR2 * (mdr - 2) +
    d6 * d * 0.0001984126984 * invR * (mdr - 1) +
    d6 * d2 * 0.0000248015873 * mdr
);

real c10Deriv = invR10 - expTerm * (
    invR10 * (mdr - 10) +
    d * invR9 * (mdr -9 ) +
    d2 * 0.5 * invR8 * (mdr - 8) +
    d3 * 0.1666666667 * invR7 * (mdr - 7) +
    d2 * d2 * 0.04166666667 * invR6 * (mdr - 6) +
    d3 * d2 * 0.008333333333 * invR5 * (mdr - 5) +
    d6 * 0.001388888889 * invR4 * (mdr - 4) +
    d6 * d * 0.0001984126984 * invR3 * (mdr - 3) +
    d6 * d2 * 0.0000248015873 * invR2 * (mdr - 2) +
    d6 * d2 * d * 0.000002755731922 * invR * (mdr - 1) +
    d6 * d2 * d2 * 0.0000002755731922 * mdr
);

real c6E = invR6 - expTerm * (
    invR6 +
    d * invR5 +
    d2 * 0.5 * invR4 +
    d3 * 0.1666666667 * invR3 +
    d2 * 0.04166666667 * invR2 +
    d3 * d2 * 0.008333333333 * invR +
    d6 * 0.001388888889
);

real c8E = invR8 - expTerm * (
    invR8 +
    d * invR7 +
    d2 * 0.5 * invR6 +
    d3 * 0.1666666667 * invR5 +
    d2 * d2 * 0.04166666667 * invR4 +
    d3 * d2 * 0.008333333333 * invR3 +
    d6 * 0.001388888889 * invR2 +
    d6 * d * 0.0001984126984 * invR +
    d6 * d2 * 0.0000248015873
);

real c10E = invR10 - expTerm * (
    invR10 +
    d * invR9 +
    d2 * 0.5 * invR8 +
    d3 * 0.1666666667 * invR7 +
    d2 * d2 * 0.04166666667 * invR6 +
    d3 * d2 * 0.008333333333 * invR5 +
    d6 * 0.001388888889 * invR4 +
    d6 * d * 0.0001984126984 * invR3 +
    d6 * d2 * 0.0000248015873 * invR2 +
    d6 * d2 * d * 0.000002755731922 * invR +
    d6 * d2 * d2 * 0.0000002755731922
);

//const real rvdw = 0.376243f;
#if USE_EWALD
    bool needCorrection = hasExclusions && isExcluded && atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS;
    unsigned int includeInteraction = ((!isExcluded && r2 < CUTOFF_SQUARED) || needCorrection);
    const real alphaR = EWALD_ALPHA*r;
    const real expAlphaRSqr = EXP(-alphaR*alphaR);
    const real prefactor = 138.935456f*posq1.w*posq2.w*invR;

#ifdef USE_DOUBLE_PRECISION
    const real erfcAlphaR = erfc(alphaR);
#else
    // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
    // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
    // error of 1.5e-7.

    const real t = RECIP(1.0f+0.3275911f*alphaR);
    const real erfcAlphaR = (0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*expAlphaRSqr;
#endif
    real tempForce = 0.0f;
#if HAS_LENNARD_JONES
    // The multiplicative term to correct for the multiplicative terms that are always
    // present in reciprocal space.  The real terms have an additive contribution
    // added in, but for excluded terms the multiplicative term is just subtracted.
    // These factors are needed in both clauses of the needCorrection statement, so
    // I declare them up here.
    #if DO_LJPME
        const real dispersionAlphaR = EWALD_DISPERSION_ALPHA*r;
        const real dar2 = dispersionAlphaR*dispersionAlphaR;
        const real dar4 = dar2*dar2;
        const real dar6 = dar4*dar2;
        const real expDar2 = EXP(-dar2);
        const real c6 = cACoefficients1.x * cACoefficients2.x;
        const real coef = invR6*c6;
        const real eprefac = 1.0f + dar2 + 0.5f*dar4;
        const real dprefac = eprefac + dar6/6.0f;
    #endif
#endif
    if (needCorrection) {
        // Subtract off the part of this interaction that was included in the reciprocal space contribution.

        if (1-erfcAlphaR > 1e-6) {
            real erfAlphaR = ERF(alphaR); // Our erfc approximation is not accurate enough when r is very small, which happens with Drude particles.
            tempForce = -prefactor*(erfAlphaR-alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
            tempEnergy += -prefactor*erfAlphaR;
        }
        else {
            includeInteraction = false;
            tempEnergy -= TWO_OVER_SQRT_PI*EWALD_ALPHA*138.935456f*posq1.w*posq2.w;
        }
#if HAS_LENNARD_JONES
        #if DO_LJPME
            // The multiplicative grid term
            tempEnergy += coef*(1.0f - expDar2*eprefac);
            tempForce += 6.0f*coef*(1.0f - expDar2*dprefac);
        #endif
#endif
    }
    else {
#if HAS_LENNARD_JONES
        // real sig = sigmaEpsilon1.x + sigmaEpsilon2.x;
        // real sig2 = invR*sig;
        // sig2 *= sig2;
        // real sig6 = sig2*sig2*sig2;
        // real eps = sigmaEpsilon1.y*sigmaEpsilon2.y;
        // real epssig6 = sig6*eps;

        real c6 = cACoefficients1.x * cACoefficients2.x;
        real c8 = cACoefficients1.y * cACoefficients2.y;
        real c10 = cBCoefficients1.x * cBCoefficients2.x;
        real c12 = cBCoefficients1.y * cBCoefficients2.y * invR6 * invR6;

	    real combinedA;
        real combinedB = buckingham1.y * buckingham2.y;

        if (combinedB == 0.0f) {
            combinedB = 0.0f;
            combinedA = 0.0f;
        } else {
            combinedB = combinedB / (buckingham1.y + buckingham2.y);
            combinedA = pow(buckingham1.x * buckingham1.y, 1.0f/buckingham1.y) * pow(buckingham2.x * buckingham2.y, 1.0f/buckingham2.y);
            combinedA = pow(combinedA, combinedB) / (2.0f * combinedB);
        }
        //	printf("SEC, B: %f, A: %f\n", combinedB, combinedA);
        real buckinghamExp = -2.0f * combinedB * r;
        real buckinghamRepulsion = combinedA * EXP(buckinghamExp);

    	// real rvdw = sigmaEpsilon1.y + sigmaEpsilon2.y;
    	// real rvdw2 = rvdw*rvdw;
    	// real rvdw6 = rvdw2*rvdw2*rvdw2;

    	// real c6D = 1.0/(r6 + rvdw6);
    	// real c8D = 1.0/(r6*r2 + rvdw6*rvdw2);
    	// real c10D = 1.0/(r6*r2*r2 + rvdw6*rvdw2*rvdw2);

        // tempForce = buckinghamRepulsion * combinedB * r + 12.0f * c12 - 6.0f * c6 - 8.0f * c8 - 10.0f * c10;
        // real ljEnergy = buckinghamRepulsion + c12 - c6 - c8 - c10;
        // ======== Tang Toennis start ========
        // 1, 1, 0.5, 0.1666666667, 0.04166666667
        // 0.008333333333, 0.001388888889, 0.0001984126984, 0.0000248015873
        // 0.000002755731922, 0.0000002755731922 
        

    	tempForce = buckinghamRepulsion * combinedB * r + 12.0f * c12 - c6Deriv * c6 - c8Deriv * c8 - c10Deriv * c10;
    	real ljEnergy = buckinghamRepulsion + c12 - c6E * c6 - c8E * c8 -c10E * c10;

        #if USE_LJ_SWITCH
        if (r > LJ_SWITCH_CUTOFF) {
            real x = r-LJ_SWITCH_CUTOFF;
            real switchValue = 1+x*x*x*(LJ_SWITCH_C3+x*(LJ_SWITCH_C4+x*LJ_SWITCH_C5));
            real switchDeriv = x*x*(3*LJ_SWITCH_C3+x*(4*LJ_SWITCH_C4+x*5*LJ_SWITCH_C5));
            tempForce = tempForce*switchValue - ljEnergy*switchDeriv*r;
            ljEnergy *= switchValue;
        }
        #endif
#if DO_LJPME
        // The multiplicative grid term
        ljEnergy += coef*(1.0f - expDar2*eprefac);
        tempForce += 6.0f*coef*(1.0f - expDar2*dprefac);
        // The potential shift accounts for the step at the cutoff introduced by the
        // transition from additive to multiplicative combintion rules and is only
        // needed for the real (not excluded) terms.  By addin these terms to ljEnergy
        // instead of tempEnergy here, the includeInteraction mask is correctly applied.
        // sig2 = sig*sig;
        // sig6 = sig2*sig2*sig2*INVCUT6;
        // epssig6 = eps*sig6;

        // BUCKINGHAM - COMMENTED OUT THE POTENTIAL SHIFT

        // The additive part of the potential shift
        // ljEnergy += epssig6*(1.0f - sig6);
        // ljEnergy += buckinghamRepulsion + c12 - c6 - c8 - c10;
        // The multiplicative part of the potential shift
        // ljEnergy += MULTSHIFT6*c6;
#endif
        tempForce += prefactor*(erfcAlphaR+alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
        tempEnergy += includeInteraction ? ljEnergy + prefactor*erfcAlphaR : 0;
#else
        tempForce = prefactor*(erfcAlphaR+alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
        tempEnergy += includeInteraction ? prefactor*erfcAlphaR : 0;
#endif
    }
    dEdR += includeInteraction ? tempForce*invR*invR : 0;
#else
#ifdef USE_CUTOFF
    unsigned int includeInteraction = (!isExcluded && r2 < CUTOFF_SQUARED);
#else
    unsigned int includeInteraction = (!isExcluded);
#endif
    real tempForce = 0.0f;
  #if HAS_LENNARD_JONES
    // real sig = sigmaEpsilon1.x + sigmaEpsilon2.x;
    // real sig2 = invR*sig;
    // sig2 *= sig2;
    // real sig6 = sig2*sig2*sig2;
    // real epssig6 = sig6*(sigmaEpsilon1.y*sigmaEpsilon2.y);
    // tempForce = epssig6*(12.0f*sig6 - 6.0f);
    // real ljEnergy = includeInteraction ? epssig6*(sig6 - 1) : 0;

    real c6 = cACoefficients1.x * cACoefficients2.x;
    real c8 = cACoefficients1.y * cACoefficients2.y;
    real c10 = cBCoefficients1.x * cBCoefficients2.x;
    real c12 = cBCoefficients1.y * cBCoefficients2.y * invR6 * invR6;

    real combinedB = buckingham1.y * buckingham2.y;
    combinedB = combinedB == 0.0f ? 0.0f : combinedB / (buckingham1.y + buckingham2.y);
    real combinedA = pow(buckingham1.x * buckingham1.y, 1.0f/buckingham1.y) * pow(buckingham2.x * buckingham2.y, 1.0f/buckingham2.y);
    combinedA = pow(combinedA, combinedB) / (2.0f * combinedB);
    // printf("B: %f, A: %f\n", combinedB, combinedA);
    real buckinghamExp = -2.0f * combinedB * r;
    real buckinghamRepulsion = combinedA * EXP(buckinghamExp);


    // real rvdw = sigmaEpsilon1.y + sigmaEpsilon2.y;
    // real rvdw2 = rvdw*rvdw;
    // real rvdw6 = rvdw2*rvdw2*rvdw2;

    // real c6D = 1.0/(r6+rvdw6);
    // real c8D = 1.0/(r6*r2 + rvdw6*rvdw2);
    // real c10D = 1.0/(r6*r2*r2 + rvdw6*rvdw2*rvdw2);

//    tempForce = buckinghamRepulsion * combinedB * r + 12.0f * c12 - 6.0f * c6 - 8.0f * c8 - 10.0f * c10;
//    real ljEnergy = includeInteraction ? buckinghamRepulsion + c12 - c6 - c8 - c10 : 0;

    tempForce = buckinghamRepulsion * combinedB * r + 12.0f * c12 - c6Deriv *c6 - c8Deriv * c8 - c10Deriv * c10;
    real ljEnergy = includeInteraction ? buckinghamRepulsion + c12 - c6E * c6 - c8E *c8 -c10E * c10 : 0;


    #if USE_LJ_SWITCH
    if (r > LJ_SWITCH_CUTOFF) {
        real x = r-LJ_SWITCH_CUTOFF;
        real switchValue = 1+x*x*x*(LJ_SWITCH_C3+x*(LJ_SWITCH_C4+x*LJ_SWITCH_C5));
        real switchDeriv = x*x*(3*LJ_SWITCH_C3+x*(4*LJ_SWITCH_C4+x*5*LJ_SWITCH_C5));
        tempForce = tempForce*switchValue - ljEnergy*switchDeriv*r;
        ljEnergy *= switchValue;
    }
    #endif
    tempEnergy += ljEnergy;
  #endif
#if HAS_COULOMB
  #ifdef USE_CUTOFF
    const real prefactor = 138.935456f*posq1.w*posq2.w;
    tempForce += prefactor*(invR - 2.0f*REACTION_FIELD_K*r2);
    tempEnergy += includeInteraction ? prefactor*(invR + REACTION_FIELD_K*r2 - REACTION_FIELD_C) : 0;
  #else
    const real prefactor = 138.935456f*posq1.w*posq2.w*invR;
    tempForce += prefactor;
    tempEnergy += includeInteraction ? prefactor : 0;
  #endif
#endif
    dEdR += includeInteraction ? tempForce*invR*invR : 0;
#endif
}
