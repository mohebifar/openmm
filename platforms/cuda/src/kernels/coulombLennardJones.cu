#include "hip/hip_runtime.h"
{
const real invR2 = invR*invR;
const real invR3 = invR2*invR;
const real invR4 = invR3*invR;
const real invR5 = invR4*invR;
const real invR6 = invR5*invR;
const real invR7 = invR6*invR;
const real invR8 = invR7*invR;
const real invR9 = invR8*invR;
const real invR10 = invR9*invR;

// const real r6 = r2*r2*r2;

const real d = sigmaEpsilon1.y + sigmaEpsilon2.y;
const real d2 = d * d * 0.5f;
const real d3 = d2 * d * 0.3333333333f;
const real d4 = d3 * d * 0.25f;
const real d5 = d4 * d * 0.2f;
const real d6 = d5 * d * 0.1666666667f;
const real d7 = d6 * d * 0.1428571429f;
const real d8 = d7 * d * 0.125f;
const real d9 = d8 * d * 0.1111111111f;
const real d10 = d9 * d * 0.1f;

const real mdr = -d * r;
real expTerm = exp(mdr);

real c6Deriv = 6.0f * invR6 + expTerm * (
    invR6 * (mdr - 6.0f) +
    d * invR5  * (mdr - 5.0f) +
    d2 * invR4 * (mdr - 4.0f) +
    d3 * invR3 * (mdr - 3.0f) +
    d4 * invR2 * (mdr - 2.0f) +
    d5 * invR * (mdr - 1.0f) +
    d6 * mdr
);

real c8Deriv = 8.0f * invR8 + expTerm * (
    invR8 * (mdr - 8.0f) +
    d * invR7 * (mdr - 7.0f) +
    d2 * invR6 * (mdr - 6.0f) +
    d3 * invR5 * (mdr - 5.0f) +
    d4 * invR4 * (mdr - 4.0f) +
    d5 * invR3 * (mdr - 3.0f) +
    d6 * invR2 * (mdr - 2.0f) +
    d7 * invR * (mdr - 1.0f) +
    d8 * mdr
);

real c10Deriv = 10.0f * invR10 + expTerm * (
    invR10 * (mdr - 10.0f) +
    d * invR9 * (mdr - 9.0f) +
    d2 * invR8 * (mdr - 8.0f) +
    d3 * invR7 * (mdr - 7.0f) +
    d4 * invR6 * (mdr - 6.0f) +
    d5 * invR5 * (mdr - 5.0f) +
    d6 * invR4 * (mdr - 4.0f) +
    d7 * invR3 * (mdr - 3.0f) +
    d8 * invR2 * (mdr - 2.0f) +
    d9 * invR * (mdr - 1.0f) +
    d10 * mdr
);

real c6E = invR6 - expTerm * (
    invR6 +
    d * invR5 +
    d2 * invR4 +
    d3 * invR3 +
    d4 * invR2 +
    d5 * invR +
    d6
);

real c8E = invR8 - expTerm * (
    invR8 +
    d * invR7 +
    d2 * invR6 +
    d3 * invR5 +
    d4 * invR4 +
    d5 * invR3 +
    d6 * invR2 +
    d7 * invR +
    d8
);

real c10E = invR10 - expTerm * (
    invR10 +
    d * invR9 +
    d2 * invR8 +
    d3 * invR7 +
    d4 * invR6 +
    d5 * invR5 +
    d6 * invR4 +
    d7 * invR3 +
    d8 * invR2 +
    d9 * invR +
    d10
);

// real combinedA;
real combinedA = buckingham1.x * buckingham2.x;
real combinedB = buckingham1.y * buckingham2.y * 0.5f;

/*
if (combinedB == 0.0f) {
            combinedB = 0.0f;
            combinedA = 0.0f;
} else {
	combinedB = combinedB / (buckingham1.y + buckingham2.y);
	combinedA = pow(buckingham1.x * buckingham1.y, 1.0f/buckingham1.y) * pow(buckingham2.x * buckingham2.y, 1.0f/buckingham2.y);
	combinedA = pow(combinedA, combinedB) / (2.0f * combinedB);
}
*/

//const real rvdw = 0.376243f;
#if USE_EWALD
    bool needCorrection = hasExclusions && isExcluded && atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS;
    unsigned int includeInteraction = ((!isExcluded && r2 < CUTOFF_SQUARED) || needCorrection);
    const real alphaR = EWALD_ALPHA*r;
    const real expAlphaRSqr = EXP(-alphaR*alphaR);
    const real prefactor = 138.935456f*posq1.w*posq2.w*invR;

#ifdef USE_DOUBLE_PRECISION
    const real erfcAlphaR = erfc(alphaR);
#else
    // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
    // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
    // error of 1.5e-7.

    const real t = RECIP(1.0f+0.3275911f*alphaR);
    const real erfcAlphaR = (0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*expAlphaRSqr;
#endif
    real tempForce = 0.0f;
#if HAS_LENNARD_JONES
    // The multiplicative term to correct for the multiplicative terms that are always
    // present in reciprocal space.  The real terms have an additive contribution
    // added in, but for excluded terms the multiplicative term is just subtracted.
    // These factors are needed in both clauses of the needCorrection statement, so
    // I declare them up here.
    #if DO_LJPME
        const real dispersionAlphaR = EWALD_DISPERSION_ALPHA*r;
        const real dar2 = dispersionAlphaR*dispersionAlphaR;
        const real dar4 = dar2*dar2;
        const real dar6 = dar4*dar2;
        const real expDar2 = EXP(-dar2);
        const real c6 = cACoefficients1.x * cACoefficients2.x;
        const real coef = invR6*c6;
        const real eprefac = 1.0f + dar2 + 0.5f*dar4;
        const real dprefac = eprefac + dar6/6.0f;
    #endif
#endif
    if (needCorrection) {
        // Subtract off the part of this interaction that was included in the reciprocal space contribution.

        if (1-erfcAlphaR > 1e-6) {
            real erfAlphaR = ERF(alphaR); // Our erfc approximation is not accurate enough when r is very small, which happens with Drude particles.
            tempForce = -prefactor*(erfAlphaR-alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
            tempEnergy += -prefactor*erfAlphaR;
        }
        else {
            includeInteraction = false;
            tempEnergy -= TWO_OVER_SQRT_PI*EWALD_ALPHA*138.935456f*posq1.w*posq2.w;
        }
#if HAS_LENNARD_JONES
        #if DO_LJPME
            // The multiplicative grid term
            tempEnergy += coef*(1.0f - expDar2*eprefac);
            tempForce += 6.0f*coef*(1.0f - expDar2*dprefac);
        #endif
#endif
    }
    else {
#if HAS_LENNARD_JONES
        // real sig = sigmaEpsilon1.x + sigmaEpsilon2.x;
        // real sig2 = invR*sig;
        // sig2 *= sig2;
        // real sig6 = sig2*sig2*sig2;
        // real eps = sigmaEpsilon1.y*sigmaEpsilon2.y;
        // real epssig6 = sig6*eps;

        real c6 = cACoefficients1.x * cACoefficients2.x;
        real c8 = cACoefficients1.y * cACoefficients2.y;
        real c10 = cBCoefficients1.x * cBCoefficients2.x;
        real c12 = cBCoefficients1.y * cBCoefficients2.y * invR6 * invR6;

        real buckinghamExp = -2.0f * combinedB * r;
        real buckinghamRepulsion = combinedA * EXP(buckinghamExp);

    	// tempForce = -buckinghamExp * buckinghamRepulsion + 12.0f * c12 - 6.0f * invR6 * c6 - 8.0f * invR8 * c8 - 10.0f * invR10 * c10;
    	// real ljEnergy = buckinghamRepulsion + c12 - invR6 * c6 - invR8 * c8 - invR10 * c10;

    	tempForce = -buckinghamExp * buckinghamRepulsion + 12.0f * c12 - c6Deriv * c6 - c8Deriv * c8 - c10Deriv * c10;
    	real ljEnergy = buckinghamRepulsion + c12 - c6E * c6 - c8E * c8 -c10E * c10;

//	if (combinedB != 0.0f) {
//		printf("SEC, r: %f, Force: %f, Energy: %f, rep: %f \n", r, tempForce, ljEnergy, buckinghamRepulsion);
//	}

        #if USE_LJ_SWITCH
        if (r > LJ_SWITCH_CUTOFF) {
            real x = r-LJ_SWITCH_CUTOFF;
            real switchValue = 1+x*x*x*(LJ_SWITCH_C3+x*(LJ_SWITCH_C4+x*LJ_SWITCH_C5));
            real switchDeriv = x*x*(3*LJ_SWITCH_C3+x*(4*LJ_SWITCH_C4+x*5*LJ_SWITCH_C5));
            tempForce = tempForce*switchValue - ljEnergy*switchDeriv*r;
            ljEnergy *= switchValue;
        }
        #endif
#if DO_LJPME
        // The multiplicative grid term
        ljEnergy += coef*(1.0f - expDar2*eprefac);
        tempForce += 6.0f*coef*(1.0f - expDar2*dprefac);
        // The potential shift accounts for the step at the cutoff introduced by the
        // transition from additive to multiplicative combintion rules and is only
        // needed for the real (not excluded) terms.  By addin these terms to ljEnergy
        // instead of tempEnergy here, the includeInteraction mask is correctly applied.
        // sig2 = sig*sig;
        // sig6 = sig2*sig2*sig2*INVCUT6;
        // epssig6 = eps*sig6;

        // BUCKINGHAM - COMMENTED OUT THE POTENTIAL SHIFT

        // The additive part of the potential shift
        // ljEnergy += epssig6*(1.0f - sig6);
        // ljEnergy += buckinghamRepulsion + c12 - c6 - c8 - c10;
        // The multiplicative part of the potential shift
        // ljEnergy += MULTSHIFT6*c6;
#endif
        tempForce += prefactor*(erfcAlphaR+alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
        tempEnergy += includeInteraction ? ljEnergy + prefactor*erfcAlphaR : 0;
#else
        tempForce = prefactor*(erfcAlphaR+alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
        tempEnergy += includeInteraction ? prefactor*erfcAlphaR : 0;
#endif
    }
    dEdR += includeInteraction ? tempForce*invR*invR : 0;
#else
#ifdef USE_CUTOFF
    unsigned int includeInteraction = (!isExcluded && r2 < CUTOFF_SQUARED);
#else
    unsigned int includeInteraction = (!isExcluded);
#endif
    real tempForce = 0.0f;
  #if HAS_LENNARD_JONES
    // real sig = sigmaEpsilon1.x + sigmaEpsilon2.x;
    // real sig2 = invR*sig;
    // sig2 *= sig2;
    // real sig6 = sig2*sig2*sig2;
    // real epssig6 = sig6*(sigmaEpsilon1.y*sigmaEpsilon2.y);
    // tempForce = epssig6*(12.0f*sig6 - 6.0f);
    // real ljEnergy = includeInteraction ? epssig6*(sig6 - 1) : 0;

    real c6 = cACoefficients1.x * cACoefficients2.x;
    real c8 = cACoefficients1.y * cACoefficients2.y;
    real c10 = cBCoefficients1.x * cBCoefficients2.x;
    real c12 = cBCoefficients1.y * cBCoefficients2.y * invR6 * invR6;

    // printf("B: %f, A: %f\n", combinedB, combinedA);
    real buckinghamExp = -2.0f * combinedB * r;
    real buckinghamRepulsion = combinedA * EXP(buckinghamExp);

    // tempForce = -buckinghamExp * buckinghamRepulsion + 12.0f * c12 - 6.0f * invR6 * c6 - 8.0f * invR8 * c8 - 10.0f * invR10 * c10;
    // real ljEnergy = includeInteraction ? buckinghamRepulsion + c12 - invR6 * c6 - invR8 * c8 - invR10 * c10 : 0;

    tempForce = -buckinghamExp * buckinghamRepulsion + 12.0f * c12 - c6Deriv *c6 - c8Deriv * c8 - c10Deriv * c10;
    real ljEnergy = includeInteraction ? buckinghamRepulsion + c12 - c6E * c6 - c8E *c8 -c10E * c10 : 0;


    #if USE_LJ_SWITCH
    if (r > LJ_SWITCH_CUTOFF) {
        real x = r-LJ_SWITCH_CUTOFF;
        real switchValue = 1+x*x*x*(LJ_SWITCH_C3+x*(LJ_SWITCH_C4+x*LJ_SWITCH_C5));
        real switchDeriv = x*x*(3*LJ_SWITCH_C3+x*(4*LJ_SWITCH_C4+x*5*LJ_SWITCH_C5));
        tempForce = tempForce*switchValue - ljEnergy*switchDeriv*r;
        ljEnergy *= switchValue;
    }
    #endif
    tempEnergy += ljEnergy;
  #endif
#if HAS_COULOMB
  #ifdef USE_CUTOFF
    const real prefactor = 138.935456f*posq1.w*posq2.w;
    tempForce += prefactor*(invR - 2.0f*REACTION_FIELD_K*r2);
    tempEnergy += includeInteraction ? prefactor*(invR + REACTION_FIELD_K*r2 - REACTION_FIELD_C) : 0;
  #else
    const real prefactor = 138.935456f*posq1.w*posq2.w*invR;
    tempForce += prefactor;
    tempEnergy += includeInteraction ? prefactor : 0;
  #endif
#endif
    dEdR += includeInteraction ? tempForce*invR*invR : 0;
#endif
}
